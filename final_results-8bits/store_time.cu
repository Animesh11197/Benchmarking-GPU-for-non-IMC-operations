
#include <stdio.h>
#include <sys/time.h>

const int threads_per_block = 32;


// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, char *name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
	printf("%s: %.5f sec\n", name, ((float) (end_time - start_time)) / (1000 * 1000));
	return end_time - start_time;
}


// Prints the specified message and quits
void die(char *message) {
	printf("%s\n", message);
	exit(1);
}

int main(int argc, char **argv) {

int N = 1000000000;


char A;
long long vector_start_time = start_timer();
for (int i = 0; i < N; i++) 
{
		A=0;
	}
stop_timer(vector_start_time, "Store Time: ");




char *A_GPU;
	if (hipMalloc((void **) &A_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
return 0;
}